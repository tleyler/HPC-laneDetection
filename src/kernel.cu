#include "hip/hip_runtime.h"
﻿#include <opencv2/highgui/highgui.hpp>
#include "opencv2/imgproc/imgproc.hpp"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <vector>
#define CHANNELS 3


__global__ void grayscaleKernel(unsigned char* rgbInput, unsigned char* grayOutput, int width, int height, int colorWidthStep, int grayWidthStep)
{
    const int xIdx = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIdx = blockIdx.y * blockDim.y + threadIdx.y;

    if (xIdx < width && yIdx < height)
    {
        const int colorOffset = yIdx * colorWidthStep + (CHANNELS * xIdx);
        const int grayOffset = yIdx * grayWidthStep + xIdx;

        const unsigned char b = rgbInput[colorOffset];
        const unsigned char g = rgbInput[colorOffset + 1];
        const unsigned char r = rgbInput[colorOffset + 2];

        const float gray = r * 0.3f + g * 0.59f + b * 0.11f;
        grayOutput[grayOffset] = static_cast<unsigned char>(gray);
    }
}

void grayscaleCuda(const cv::Mat& hostInput, cv::Mat& hostOutput)
{
    // Allocate memory on device for input and output
    unsigned char* deviceInput;
    unsigned char* deviceOutput;
    const int colorBytes = hostInput.step * hostInput.rows;
    const int grayBytes = hostOutput.step * hostOutput.rows;
    hipMalloc<unsigned char>(&deviceInput, colorBytes);
    hipMalloc<unsigned char>(&deviceOutput, grayBytes);

    // Copy memory from host to device
    hipMemcpy(deviceInput, hostInput.ptr(), colorBytes, hipMemcpyHostToDevice);

    // Call the kernel to convert the image to grayscale
    const dim3 blockSize(16, 16, 1);
    const dim3 gridSize((hostInput.cols + blockSize.x - 1) / blockSize.x, (hostInput.rows + blockSize.y - 1) / blockSize.y, 1);
    grayscaleKernel << <gridSize, blockSize >> > (deviceInput, deviceOutput, hostInput.cols, hostInput.rows, hostInput.step, hostOutput.step);

    // Copy memory back to host after kernel is complete
    hipDeviceSynchronize();
    hipMemcpy(hostOutput.ptr(), deviceOutput, grayBytes, hipMemcpyDeviceToHost);
    hipFree(deviceInput);
    hipFree(deviceOutput);
}

// This function takes in a path to a video file (which are passed in as command line args to main)
// as the first parameter and outputs each extracted frame to a vector of Mat items which is passed 
// in as the second parameter to the function. 
//extern "C++"
void extractFrames(const std::string& videoFilePath, std::vector<cv::Mat>& framesOut)
{
    try
    {
        cv::VideoCapture cap(videoFilePath);
        if (!cap.isOpened())
        {
            std::cerr << "Unable to open video file!" << std::endl;
            return;
        }
        for (int frameNum = 0; frameNum < cap.get(cv::CAP_PROP_FRAME_COUNT); frameNum++)
        {
            cv::Mat frame;
            cap >> frame;
            framesOut.push_back(frame);
            // VISUAL DEBUG: display each frame on screen
            // cv::imshow("Extracted Frame", frame);
            // waitKey(0);
        }
    }
    catch (cv::Exception& e)
    {
        std::cerr << e.msg << std::endl;
    }
}


// This function accepts a single frame and detects edges in it using opencv
// Canny(). It returns the edge detected image.
//extern "C++"
cv::Mat opencvCanny(const cv::Mat& frame) {
    // changing this number effects the amount of edges that it detects. The 
    // larger the number is, the less it will detect, only picking up larger 
    // edges. Through some quick experimentation I settled on 75, but this can
    // be adjusted later if we need more or less edges.
    double edgeThreshold = 75.0;

    // this mat will hold the edges image
    cv::Mat edgeDetectedFrame;
    cv::Canny(frame, edgeDetectedFrame, edgeThreshold, edgeThreshold * 3.0, 3);

    return edgeDetectedFrame;
}

// COMMAND LINE ARGUMENTS
// argv[0] = program name
// argv[1] = file path to video file
//extern "C++"
int main(int argc, char** argv)
{
    std::string videoFilePath = argv[1];
    std::vector<cv::Mat> framesOutput;
    extractFrames(videoFilePath, framesOutput);

    for (int i = 0; i < framesOutput.size(); i++)
    {
        cv::Mat image = framesOutput[i];
        const int rows = image.rows;
        const int cols = image.cols;
        //imshow("Extracted Frame", image);
        //cv::waitKey(0);

        // convert the image to grayscale 
        cv::Mat grayscale = cv::Mat(rows, cols, CV_8UC1);
        grayscaleCuda(image, grayscale);
        imshow("Grayscale Image", grayscale);
        cv::waitKey(0);

        // Apply and output opencvCanny() to each extracted frame
        //imshow("Edge Detected Frame", opencvCanny(image));
        //waitKey(0);
    }

    return 0;
}